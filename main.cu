#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <array>
#include <vector>
#include <chrono>
#include <string>
#include <fstream>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLEW_STATIC
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <cstdlib>

#define GLM_FORCE_CUDA
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"

#include "shader.hpp"

namespace cuda_function
{
    template<typename T>
    __global__ void vector_add(T* a,T* b,T* c,size_t n)
    {
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n) {
            c[idx] = a[idx] + b[idx];
        }
    }
}

template<typename T>
class TestClass {
public:
    TestClass()
    {

    }
    TestClass(std::vector<T> v) : v_(v)
    {

    }
    std::vector<T> add(std::vector<T> v)
    {
        assert(v_.size() == v.size());
        auto start_time = std::chrono::system_clock::now();
        T* d_a;
        T* d_b;
        T* d_c;
        size_t bytes = v_.size() * sizeof(T);
        hipMalloc(&d_a, bytes);
        hipMalloc(&d_b, bytes);
        hipMalloc(&d_c, bytes);
        hipMemcpy(d_a, v_.data(), bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, v.data(), bytes, hipMemcpyHostToDevice);
        size_t block_size = 32;
        dim3 dim_block(block_size);
        dim3 dim_grid(ceil(v_.size() / (float)block_size));
        cuda_function::vector_add <<<dim_grid, dim_block >>> (d_a, d_b, d_c, v_.size());
        hipDeviceSynchronize();
        T* h_c = (T*)malloc(bytes);
        hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
        std::vector<T> ret;
        ret.reserve(v_.size());
        for (size_t i = 0; i < v_.size(); i++) {
            ret.push_back(h_c[i]);
        }
        auto end_time = std::chrono::system_clock::now();
        float elapsed_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
        printf("elapsed time = %f[ms]\n", elapsed_time);
        return ret;
    }
private:
    std::vector<T> v_;
};

static const int WIN_WIDTH = 500;                 // �E�B���h�E�̕�
static const int WIN_HEIGHT = 500;                 // �E�B���h�E�̍���
static const std::string WIN_TITLE = "OpenGL Course";     // �E�B���h�E�̃^�C�g��

int main() 
{
    // OpenGL������������
    if (glfwInit() == GL_FALSE) {
        std::cerr << "Can't initialize GLFW" << std::endl;
        return 1;
    }

    atexit(glfwTerminate);

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    // Window�̍쐬
    GLFWwindow* window = glfwCreateWindow(WIN_WIDTH, WIN_HEIGHT, WIN_TITLE.c_str(),NULL, NULL);
    if (window == NULL) {
        std::cerr << "Can't create GLFW window." << std::endl;
        return 1;
    }

    // OpenGL�̕`��Ώۂ�Window��ǉ�
    glfwMakeContextCurrent(window);

    glewExperimental = GL_TRUE;
    if (glewInit() != GLEW_OK)
    {
        // GLEW �̏������Ɏ��s����
        std::cerr << "Can't initialize GLEW" << std::endl;
        return 1;
    }

    glfwSwapInterval(1);

    // ������
    GLint shader = makeShader("shader.vert", "shader.frag");
    // 2���̎O�p�|���S��
    std::vector<glm::vec3> positionList = {
        glm::vec3(0, 0, 1),glm::vec3(1,0, 0),glm::vec3(0, 0, 0),
        glm::vec3(0, 0, 1),glm::vec3(0, 0, 0),glm::vec3(0, 1, 0),
    };
    // attribute ���w�肷��
    GLint positionLocation = glGetAttribLocation(shader, "position");
    // ���_�o�b�t�@�I�u�W�F�N�g���쐬
    GLuint positionBuffer;
    glGenBuffers(1, &positionBuffer);
    // GPU���ɒ��_�o�b�t�@�I�u�W�F�N�g�Ƀ������̈���m�ۂ���
    glBindBuffer(GL_ARRAY_BUFFER, positionBuffer);
    glBufferData(GL_ARRAY_BUFFER, sizeof(glm::vec3) * positionList.size(), positionList.data(), GL_STATIC_DRAW);

    GLuint matrixID = glGetUniformLocation(shader, "MVP");

    // ���C�����[�v
    while (glfwWindowShouldClose(window) == GL_FALSE) 
    {
        // �`��
        glUseProgram(shader);

        glEnable(GL_DEPTH_TEST);
        glDepthFunc(GL_LESS);
        glClearColor(0.2f, 0.2f, 0.2f, 0.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        // �錾���ɂ͒P�ʍs�񂪓����Ă���
        glm::mat4 modelMat, viewMat, projectionMat;

        // View�s����v�Z
        viewMat = glm::lookAt(
            glm::vec3(2.0, 2.0, 2.0), // ���[���h��Ԃł̃J�����̍��W
            glm::vec3(0.0, 0.0, 0.0), // ���Ă���ʒu�̍��W
            glm::vec3(0.0, 0.0, 1.0)  // ������������B(0,1.0,0)�ɐݒ肷���y������ɂȂ�܂�
        );

        // Projection�s����v�Z
        projectionMat = glm::perspective(
            glm::radians(45.0f), // �Y�[���̓x����(�ʏ�90�`30)
            (GLfloat)WIN_WIDTH / (GLfloat)WIN_HEIGHT,		// �A�X�y�N�g��
            0.1f,		// �߂��̃N���b�s���O����
            100.0f		// �����̃N���b�s���O����
        );

        // ModelViewProjection�s����v�Z
        glm::mat4 mvpMat = projectionMat * viewMat * modelMat;

        // ���݃o�C���h���Ă���V�F�[�_��uniform�ϐ�"MVP"�ɕϊ��s��𑗂�
        // 4�ڂ̈����͍s��̍ŏ��̃A�h���X��n���Ă��܂��B
        glUniformMatrix4fv(matrixID, 1, GL_FALSE, &mvpMat[0][0]);

        // positionLocation�Ŏw�肳�ꂽattribute��L����
        glEnableVertexAttribArray(positionLocation);
        // positionBuffer�Ƀo�C���h
        glBindBuffer(GL_ARRAY_BUFFER, positionBuffer);
        // attribute�ϐ�position�Ɋ��蓖�Ă�
        // GPU���������ɑ����Ă������f�[�^���o�[�e�b�N�X�V�F�[�_�[�Ŏg���w��ł�
        glVertexAttribPointer(positionLocation, 3, GL_FLOAT, GL_FALSE, 0, (void*)0);

        glDrawArrays(GL_TRIANGLES, 0, 6);

        // �`��p�o�b�t�@�̐؂�ւ�
        glfwSwapBuffers(window);
        glfwPollEvents();
    }
    return 0;
}